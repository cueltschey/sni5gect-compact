#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 256
#include "shadower/hdr/ssb_cuda.cuh"
#include <complex>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <vector>

// Kernel for complex conjugate multiplication in the frequency domain
__global__ void complex_conj_mult(hipfftComplex* input, hipfftComplex* pss_seq, hipfftComplex* output, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    hipfftComplex a = input[idx];
    hipfftComplex b = pss_seq[idx]; // Assume pre-conjugated
    b.y            = -b.y;
    output[idx].x  = a.x * b.x - a.y * b.y;
    output[idx].y  = a.x * b.y + a.y * b.x;
  }
}

// Kernel to compute the absolute squared magnitude (correlation power)
__global__ void compute_power(hipfftComplex* input, float* power, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    power[idx] = input[idx].x * input[idx].x + input[idx].y * input[idx].y; // |z|^2
  }
}

// Kernel to normalize the correlation
__global__ void normalize_correlation(float* corr, float* power, int N, float scale_factor)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N && power[idx] > 0) {
    corr[idx] /= (power[idx] * scale_factor);
  }
}

__global__ void find_max_kernel(float* d_data, int size, float* d_max_val, int* d_max_idx)
{
  extern __shared__ float shared_data[];
  int*                    shared_idx = (int*)&shared_data[blockDim.x];

  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  // Load data into shared memory
  if (idx < size) {
    shared_data[tid] = d_data[idx];
    shared_idx[tid]  = idx;
  } else {
    shared_data[tid] = -1e10f; // Very small value for comparison
    shared_idx[tid]  = -1;
  }
  __syncthreads();

  // Perform reduction to find max value
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (tid < stride) {
      if (shared_data[tid] < shared_data[tid + stride]) {
        shared_data[tid] = shared_data[tid + stride];
        shared_idx[tid]  = shared_idx[tid + stride];
      }
    }
    __syncthreads();
  }

  // Write the result of this block to global memory
  if (tid == 0) {
    d_max_val[blockIdx.x] = shared_data[0];
    d_max_idx[blockIdx.x] = shared_idx[0];
  }
}

void SSBCuda::find_max(float* d_data, int size, float* max_val, int* max_idx)
{
  // clang-format off
  find_max_kernel<<<compareBlocksPerGrid, THREADS_PER_BLOCK, THREADS_PER_BLOCK * (sizeof(float) + sizeof(int))>>>(d_data, size, d_block_max_vals, d_block_max_idxs);
  // clang-format on
  hipMemcpyAsync(
      h_block_max_vals, d_block_max_vals, compareBlocksPerGrid * sizeof(float), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(
      h_block_max_idxs, d_block_max_idxs, compareBlocksPerGrid * sizeof(int), hipMemcpyDeviceToHost, stream);
  // Final reduction on CPU
  *max_val = h_block_max_vals[0];
  *max_idx = h_block_max_idxs[0];
  for (int i = 1; i < compareBlocksPerGrid; i++) {
    if (h_block_max_vals[i] > *max_val) {
      *max_val = h_block_max_vals[i];
      *max_idx = h_block_max_idxs[i];
    }
  }
}

SSBCuda::SSBCuda(double                      srate_,
                 double                      dl_freq_,
                 double                      ssb_freq_,
                 srsran_subcarrier_spacing_t scs_,
                 srsran_ssb_pattern_t        pattern_,
                 srsran_duplex_mode_t        duplex_mode_) :
  srate(srate_), dl_freq(dl_freq_), ssb_freq(ssb_freq_), scs(scs_), pattern(pattern_), duplex_mode(duplex_mode_)
{
}

SSBCuda::~SSBCuda() {}

void SSBCuda::cleanup()
{
  if (d_freq) {
    hipFree(d_freq);
  }
  if (d_corr) {
    hipFree(d_corr);
  }
  if (d_pss_seq) {
    hipFree(d_pss_seq);
  }
  if (d_corr_mag) {
    hipFree(d_corr_mag);
  }
  if (d_power) {
    hipFree(d_power);
  }
  if (fft_plan) {
    hipfftDestroy(fft_plan);
  }
}

bool SSBCuda::init(uint32_t N_id_2)
{
  srsran_ssb_args_t ssb_args = {};
  ssb_args.max_srate_hz      = srate;
  ssb_args.min_scs           = scs;
  ssb_args.enable_search     = true;
  ssb_args.enable_measure    = true;
  ssb_args.enable_decode     = true;
  if (srsran_ssb_init(&ssb, &ssb_args) != 0) {
    printf("Error initialize ssb\n");
    return false;
  }
  srsran_ssb_cfg_t ssb_cfg = {};
  ssb_cfg.srate_hz         = srate;
  ssb_cfg.center_freq_hz   = dl_freq;
  ssb_cfg.ssb_freq_hz      = ssb_freq;
  ssb_cfg.scs              = scs;
  ssb_cfg.pattern          = pattern;
  ssb_cfg.duplex_mode      = duplex_mode;
  ssb_cfg.periodicity_ms   = 10;
  if (srsran_ssb_set_cfg(&ssb, &ssb_cfg) < SRSRAN_SUCCESS) {
    printf("Error set srsran_ssb_set_cfg\n");
    return false;
  }

  hipfftPlan1d(&fft_plan, ssb.corr_sz, HIPFFT_C2C, 1);
  hipHostMalloc((void**)&h_pin_time, (ssb.sf_sz + ssb.ssb_sz) * sizeof(hipfftComplex));
  hipMalloc((void**)&d_freq, ssb.corr_sz * sizeof(hipfftComplex));
  hipMalloc((void**)&d_time, ssb.corr_sz * sizeof(hipfftComplex));
  hipMalloc((void**)&d_corr, ssb.corr_sz * sizeof(hipfftComplex));
  hipMalloc((void**)&d_pss_seq, ssb.corr_sz * sizeof(hipfftComplex));
  hipMalloc((void**)&d_corr_mag, ssb.corr_window * sizeof(float));
  hipMalloc((void**)&d_power, ssb.corr_sz * sizeof(float));
  hipMemcpy(d_pss_seq, ssb.pss_seq[N_id_2], ssb.corr_sz * sizeof(hipfftComplex), hipMemcpyHostToDevice);
  compareBlocksPerGrid = (ssb.corr_window + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipMalloc((void**)&d_block_max_vals, compareBlocksPerGrid * sizeof(float));
  hipMalloc((void**)&d_block_max_idxs, compareBlocksPerGrid * sizeof(int));
  h_block_max_vals = (float*)malloc(compareBlocksPerGrid * sizeof(float));
  h_block_max_idxs = (int*)malloc(compareBlocksPerGrid * sizeof(int));
  hipStreamCreate(&stream);
  hipfftSetStream(fft_plan, stream);
  return true;
}

int SSBCuda::ssb_pss_find_cuda(cf_t* in, uint32_t nof_samples, uint32_t* found_delay)
{
  if (ssb.corr_sz == 0) {
    return -1;
  }
  uint32_t best_delay = 0;
  float    best_corr  = 0;
  uint32_t t_offset   = 0;
  uint32_t total_len  = nof_samples + ssb.ssb_sz;
  memcpy(h_pin_time, h_pin_time + ssb.sf_sz, sizeof(hipfftComplex) * ssb.ssb_sz);
  memcpy(h_pin_time + ssb.ssb_sz, in, sizeof(hipfftComplex) * nof_samples);
  while ((t_offset + ssb.symbol_sz) < total_len) {
    // Number of samples taken in this iteration
    uint32_t chunk_size = ssb.corr_sz;

    // Detect if the correlation input exceeds the input length, take the maximum amount of samples
    if (t_offset + ssb.corr_sz > total_len) {
      chunk_size = total_len - t_offset;
    }

    // Copy the amount of samples
    hipMemcpyAsync(d_time, h_pin_time + t_offset, sizeof(hipfftComplex) * chunk_size, hipMemcpyHostToDevice, stream);

    // Append zeros if there's space left
    if (chunk_size < ssb.corr_sz) {
      hipMemsetAsync(d_time + chunk_size, 0, sizeof(hipfftComplex) * (ssb.corr_sz - chunk_size), stream);
    }

    // Perform the FFT covnert to frequncy domain
    hipfftExecC2C(fft_plan, d_time, d_freq, HIPFFT_FORWARD);

    // Perform correlation between frequency domain and PSS sequence
    int blocks = (ssb.corr_sz + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // clang-format off
    complex_conj_mult<<<blocks, THREADS_PER_BLOCK>>>(d_freq, d_pss_seq, d_corr, ssb.corr_sz);
    // clang-format on

    hipStreamSynchronize(stream);

    hipfftExecC2C(fft_plan, d_corr, d_corr, HIPFFT_BACKWARD);

    hipStreamSynchronize(stream);

    // clang-format off
    compute_power<<<blocks, THREADS_PER_BLOCK>>>(d_corr, d_corr_mag, ssb.corr_window);
    hipStreamSynchronize(stream);
    // clang-format on 

    float peak_val;
    int peak_idx;
    find_max(d_corr_mag, ssb.corr_window, &peak_val, &peak_idx);

    if (best_corr < peak_val) {
      best_corr  = peak_val;
      best_delay = peak_idx + t_offset;
    }
    t_offset += ssb.corr_window;
  }
  *found_delay = best_delay;
  return 0;
}