#include "hip/hip_runtime.h"
#define THREADS_PER_BLOCK 256
#include "shadower/hdr/ssb_cuda.cuh"
#include <chrono>
#include <complex>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <vector>

// Kernel for complex conjugate multiplication in the frequency domain
__global__ void
complex_conj_mult_slide_window(hipfftComplex* input, hipfftComplex* pss_seq, hipfftComplex* output, uint32_t pss_size)
{
  int tidx        = threadIdx.x;
  int seg_idx     = blockIdx.y;
  int element_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idx         = seg_idx * pss_size + element_idx;

  if (element_idx < pss_size) {
    hipfftComplex e   = input[idx];
    hipfftComplex pss = pss_seq[element_idx];
    pss.y            = -pss.y;
    output[idx].x    = e.x * pss.x - e.y * pss.y;
    output[idx].y    = e.x * pss.y + e.y * pss.x;
  }
}

// Kernel to compute the absolute squared magnitude (correlation power)
__global__ void compute_power(hipfftComplex* input, float* power, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    power[idx] = input[idx].x * input[idx].x + input[idx].y * input[idx].y; // |z|^2
  }
}

__global__ void find_max_kernel(float* d_data, int size, float* d_max_val, int* d_max_idx)
{
  __shared__ float shared_data[1024];
  __shared__ float shared_idx[1024];

  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  // Load data into shared memory
  if (idx < size) {
    shared_data[tid] = d_data[idx];
    shared_idx[tid]  = idx;
  } else {
    shared_data[tid] = -1e10f; // Very small value for comparison
    shared_idx[tid]  = -1;
  }
  __syncthreads();

  // Perform reduction to find max value
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (tid < stride) {
      if (shared_data[tid] < shared_data[tid + stride]) {
        shared_data[tid] = shared_data[tid + stride];
        shared_idx[tid]  = shared_idx[tid + stride];
      }
    }
    __syncthreads();
  }

  // Write the result of this block to global memory
  if (tid == 0) {
    d_max_val[blockIdx.x] = shared_data[0];
    d_max_idx[blockIdx.x] = shared_idx[0];
  }
}

void SSBCuda::find_max(float* d_data, int size, float* max_val, int* max_idx)
{
  // clang-format off
  find_max_kernel<<<compareBlocksPerGrid, THREADS_PER_BLOCK, THREADS_PER_BLOCK * (sizeof(float) + sizeof(int))>>>(d_data, size, d_block_max_vals, d_block_max_idxs);
  // clang-format on
  hipMemcpyAsync(
      h_block_max_vals, d_block_max_vals, compareBlocksPerGrid * sizeof(float), hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(
      h_block_max_idxs, d_block_max_idxs, compareBlocksPerGrid * sizeof(int), hipMemcpyDeviceToHost, stream);
  // Final reduction on CPU
  *max_val = h_block_max_vals[0];
  *max_idx = h_block_max_idxs[0];
  for (int i = 1; i < compareBlocksPerGrid; i++) {
    if (h_block_max_vals[i] > *max_val) {
      *max_val = h_block_max_vals[i];
      *max_idx = h_block_max_idxs[i];
    }
  }
}

SSBCuda::SSBCuda(double                      srate_,
                 double                      dl_freq_,
                 double                      ssb_freq_,
                 srsran_subcarrier_spacing_t scs_,
                 srsran_ssb_pattern_t        pattern_,
                 srsran_duplex_mode_t        duplex_mode_) :
  srate(srate_), dl_freq(dl_freq_), ssb_freq(ssb_freq_), scs(scs_), pattern(pattern_), duplex_mode(duplex_mode_)
{
}

SSBCuda::~SSBCuda() {}

void SSBCuda::cleanup()
{
  if (h_pin_time) {
    hipHostFree(h_pin_time);
  }
  if (d_time) {
    hipFree(d_time);
  }
  if (d_freq) {
    hipFree(d_freq);
  }
  if (d_corr) {
    hipFree(d_corr);
  }
  if (d_pss_seq) {
    hipFree(d_pss_seq);
  }
  if (d_corr_mag) {
    hipFree(d_corr_mag);
  }
  if (fft_plan) {
    hipfftDestroy(fft_plan);
  }
  if (d_block_max_idxs) {
    hipFree(d_block_max_idxs);
  }
  if (d_block_max_vals) {
    hipFree(d_block_max_vals);
  }
  if (h_block_max_idxs) {
    free(h_block_max_idxs);
  }
  if (h_block_max_vals) {
    free(h_block_max_vals);
  }
  if (stream) {
    hipStreamDestroy(stream);
  }
  srsran_ssb_free(&ssb);
}

bool SSBCuda::init(uint32_t N_id_2)
{
  srsran_ssb_args_t ssb_args = {};
  ssb_args.max_srate_hz      = srate;
  ssb_args.min_scs           = scs;
  ssb_args.enable_search     = true;
  ssb_args.enable_measure    = true;
  ssb_args.enable_decode     = true;
  if (srsran_ssb_init(&ssb, &ssb_args) != 0) {
    printf("Error initialize ssb\n");
    return false;
  }
  srsran_ssb_cfg_t ssb_cfg = {};
  ssb_cfg.srate_hz         = srate;
  ssb_cfg.center_freq_hz   = dl_freq;
  ssb_cfg.ssb_freq_hz      = ssb_freq;
  ssb_cfg.scs              = scs;
  ssb_cfg.pattern          = pattern;
  ssb_cfg.duplex_mode      = duplex_mode;
  ssb_cfg.periodicity_ms   = 10;
  if (srsran_ssb_set_cfg(&ssb, &ssb_cfg) < SRSRAN_SUCCESS) {
    printf("Error set srsran_ssb_set_cfg\n");
    return false;
  }

  total_len    = ssb.sf_sz + ssb.ssb_sz;
  last_len     = total_len;
  round        = (total_len + ssb.corr_window - 1) / ssb.corr_window;
  total_len    = round * ssb.corr_sz;
  int n[1]     = {(int)ssb.corr_sz};
  int embed[1] = {1};
  hipfftPlanMany(&fft_plan, 1, n, embed, 1, ssb.corr_window, embed, 1, ssb.corr_sz, HIPFFT_C2C, round);
  hipfftPlan1d(&ifft_plan, ssb.corr_sz, HIPFFT_C2C, round);
  hipHostMalloc((void**)&h_pin_time, total_len * sizeof(hipfftComplex)); // Pinned memory
  hipMalloc((void**)&d_time, total_len * sizeof(hipfftComplex));         // Time domain buffer
  hipMalloc((void**)&d_freq, total_len * sizeof(hipfftComplex));         // Frequency domain buffer
  hipMalloc((void**)&d_corr, total_len * sizeof(hipfftComplex));         // Correlation result buffer
  hipMalloc((void**)&d_pss_seq, ssb.corr_sz * sizeof(hipfftComplex));    // PSS sequence buffer
  hipMalloc((void**)&d_corr_mag, total_len * sizeof(hipfftComplex));     // Correlation magnitude buffer

  // Copy pss sequence to device
  hipMemcpy(d_pss_seq, ssb.pss_seq[N_id_2], ssb.corr_sz * sizeof(hipfftComplex), hipMemcpyHostToDevice);

  // Allocate memory for CUDA kernel to find max value
  compareBlocksPerGrid = (ssb.corr_window + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipMalloc((void**)&d_block_max_vals, compareBlocksPerGrid * sizeof(float));
  hipMalloc((void**)&d_block_max_idxs, compareBlocksPerGrid * sizeof(int));
  h_block_max_vals = (float*)malloc(compareBlocksPerGrid * sizeof(float));
  h_block_max_idxs = (int*)malloc(compareBlocksPerGrid * sizeof(int));

  // Create a CUDA stream for asynchronous data transfer
  hipStreamCreate(&stream);
  hipfftSetStream(fft_plan, stream);
  return true;
}

int SSBCuda::ssb_pss_find_cuda(cf_t* in, uint32_t nof_samples, uint32_t* found_delay)
{
  if (ssb.corr_sz == 0) {
    return -1;
  }
  /* Copy the end of last ssb_sz to current buffer */
  memcpy(h_pin_time, h_pin_time + last_len - ssb.ssb_sz, sizeof(hipfftComplex) * ssb.ssb_sz);
  /* Copy the current input buffer to pin buffer */
  memcpy(h_pin_time + ssb.ssb_sz, in, sizeof(hipfftComplex) * nof_samples);
  /* Keep tracking the total len */
  last_len = nof_samples + ssb.ssb_sz;
  /* Set the remaining buffer to zero */
  memset(h_pin_time + last_len, 0, sizeof(hipfftComplex) * (total_len - last_len));

  /* Copy the data to cuda device */
  hipMemcpyAsync(d_time, h_pin_time, sizeof(hipfftComplex) * last_len, hipMemcpyHostToDevice, stream);

  /* Convert time domain data to frequency domain */
  hipfftExecC2C(fft_plan, d_time, d_freq, HIPFFT_FORWARD);

  /* Perform correlation between frequency domain and PSS sequence */
  dim3 numBlocks((ssb.corr_sz + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, round);
  // clang-format off
  complex_conj_mult_slide_window<<<numBlocks, THREADS_PER_BLOCK>>>(d_freq, d_pss_seq, d_corr, ssb.corr_sz);
  // clang-format on

  /* Convert the frequency domain correlation to time domain */
  hipfftExecC2C(ifft_plan, d_corr, d_corr, HIPFFT_BACKWARD);

  /* Compute the power of the correlation */
  int compareNumBlocks = total_len / THREADS_PER_BLOCK;
  // clang-format off
  compute_power<<<compareNumBlocks, THREADS_PER_BLOCK>>>(d_corr, d_corr_mag, total_len);
  // clang-format on

  //   float best_corr  = 0;
  //   int   best_delay = -1;
  //   for (int r = 0; r < round; r++) {
  //     float round_max_corr  = 0;
  //     int   round_max_delay = -1;
  //     find_max(d_corr_mag + r * ssb.corr_sz, ssb.corr_window, &round_max_corr, &round_max_delay);
  //     if (round_max_corr > best_corr) {
  //       best_corr  = round_max_corr;
  //       best_delay = r * ssb.corr_window + round_max_delay;
  //     }
  //   }
  //   *found_delay = best_delay;
  //   return 0;
  // }
  float best_corr  = 0;
  int   best_delay = -1;
  find_max(d_corr_mag, total_len, &best_corr, &best_delay);
  int round_number, round_offset;
  round_number = best_delay / ssb.corr_sz;
  round_offset = best_delay % ssb.corr_sz;
  *found_delay = round_number * ssb.corr_window + round_offset;
  return 0;
}