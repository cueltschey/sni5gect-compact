#include "hip/hip_runtime.h"
#include "shadower/hdr/constants.h"
#include "shadower/hdr/fft_processor.cuh"
#include <chrono>
#include <cmath>

// Define CUDA kernel
__global__ void
gpu_vec_sc_prod_ccc(hipfftComplex* d_signal, hipfftComplex* d_phase_list, int fft_size, int symbols_per_slot)
{
  int idx        = threadIdx.x + blockIdx.x * blockDim.x;
  int symbol_idx = blockIdx.y;

  if (idx < fft_size) {
    int          index = symbol_idx * fft_size + idx;
    hipfftComplex phase = d_phase_list[symbol_idx];
    // Complex multiplication: result = d_signal * phase
    hipfftComplex result;
    result.x        = d_signal[index].x * phase.x - d_signal[index].y * phase.y;
    result.y        = d_signal[index].x * phase.y + d_signal[index].y * phase.x;
    d_signal[index] = result;
  }
}

// Function to launch the kernel
void launch_gpu_vec_sc_prod_ccc(hipfftComplex* d_signal, hipfftComplex* d_phase_list, int fft_size, int symbols_per_slot)
{
  dim3 threadsPerBlock(256);
  dim3 numBlocks((fft_size + threadsPerBlock.x - 1) / threadsPerBlock.x, symbols_per_slot);
  // clang-format off
  gpu_vec_sc_prod_ccc<<<numBlocks, threadsPerBlock>>>(d_signal, d_phase_list, fft_size, symbols_per_slot);
  // clang-format on
}

FFTProcessor::FFTProcessor(double                      sample_rate_,
                           srsran_subcarrier_spacing_t scs_,
                           uint32_t                    num_prbs_,
                           double                      center_freq) :
  sample_rate(sample_rate_),
  scs(scs_),
  two_pow_numerology(1 << scs),
  scs_khz((1 << scs) * 15),
  nof_sc(num_prbs_ * SRSRAN_NRE),
  slots_per_subframe(1 << scs),
  sf_len(sample_rate * SF_DURATION),
  symbols_per_subframe(symbols_per_slot * slots_per_subframe)
{
  fft_size  = sf_len / scs_khz;
  half_fft  = fft_size / 2;
  half_subc = nof_sc / 2;
  /* Calculate the duration in the unit of Tc */
  ofdm_units    = 2048.0 * K * 1.0 / two_pow_numerology;
  cp_units      = 144.0 * K * 1.0 / two_pow_numerology;
  long_cp_units = (144.0 + 16.0) * K * 1.0 / two_pow_numerology;

  /* Calculate the duration in seconds */
  ofdm_duration      = ofdm_units * Tc;
  normal_cp_duration = cp_units * Tc;
  long_cp_duration   = long_cp_units * Tc;

  /* Calculate the cyclic prefix length */
  ofdm_length    = std::floor(ofdm_duration * sample_rate);
  cp_length      = std::floor(normal_cp_duration * sample_rate);
  long_cp_length = std::floor(long_cp_duration * sample_rate);

  /* Initialize the cyclic prefix */
  cp_length_list.resize(symbols_per_subframe, cp_length);
  // Long CP list for 0 and 7 * 2^(miu - 1)
  cp_length_list[0]                      = long_cp_length;
  cp_length_list[7 * two_pow_numerology] = long_cp_length;

  uint32_t             count = 0;
  std::complex<double> I(0, 1);
  phase_compensation_conj_list.resize(symbols_per_subframe);
  for (uint32_t l = 0; l < symbols_per_subframe; l++) {
    uint32_t cp_len = cp_length_list[l];
    count += cp_len;
    double t_start   = (double)count / sample_rate;
    double phase_rad = -2.0 * M_PI * center_freq * t_start;

    std::complex<double> phase_comp      = std::exp(I * phase_rad);
    std::complex<double> phase_comp_conj = std::conj(phase_comp);
    phase_compensation_conj_list[l]      = phase_comp_conj;
    count += ofdm_length;
  }

  hipError_t error = hipMalloc((void**)&d_signal, symbols_per_slot * fft_size * sizeof(hipfftComplex));
  if (error != hipError_t::hipSuccess) {
    throw std::runtime_error("hipMalloc failed");
  }

  hipError_t error2 = hipHostMalloc((void**)&h_pinned_buffer, symbols_per_slot * fft_size * sizeof(hipfftComplex));
  if (error2 != hipError_t::hipSuccess) {
    throw std::runtime_error("hipHostMalloc failed");
  }
  hipfftResult result = hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, symbols_per_slot);
  if (result != HIPFFT_SUCCESS) {
    throw std::runtime_error("CUFFT error: Plan creation failed");
  }

  hipError_t error3 = hipMalloc((void**)&phase_compensation_list_gpu, symbols_per_subframe * sizeof(hipfftComplex));
  if (error3 != hipError_t::hipSuccess) {
    throw std::runtime_error("hipMalloc failed");
  }
  hipMemcpy(phase_compensation_list_gpu,
             phase_compensation_conj_list.data(),
             symbols_per_subframe * sizeof(hipfftComplex),
             hipMemcpyHostToDevice);
  hipStreamCreate(&stream);
  hipfftSetStream(plan, stream);
}

/* Process the samples from a slot at a time */
void FFTProcessor::process_samples(cf_t* buffer, cf_t* ofdm_symbols, uint32_t slot_idx)
{
  uint32_t start_idx      = slot_idx % slots_per_subframe * symbols_per_slot;
  uint32_t current_offset = 0;
  uint32_t idx, cyclic_prefix_length;

  // Use regular memcpy instead of hipMemcpyHostToHost
  for (uint32_t i = 0; i < symbols_per_slot; i++) {
    cyclic_prefix_length = cp_length_list[start_idx + i];
    current_offset += cyclic_prefix_length; // remove the cyclic prefix
    memcpy(h_pinned_buffer + i * fft_size, buffer + current_offset, sizeof(hipfftComplex) * fft_size);
    current_offset += ofdm_length; // proceeds after processing the OFDM symbol
  }

  // Asynchronous transfer to GPU
  hipMemcpyAsync(
      d_signal, h_pinned_buffer, symbols_per_slot * fft_size * sizeof(hipfftComplex), hipMemcpyHostToDevice, stream);

  // Run fft
  hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);

  // Apply phase compensation
  launch_gpu_vec_sc_prod_ccc(d_signal, phase_compensation_list_gpu, fft_size, symbols_per_slot);

  // Copy result back asynchronously
  hipMemcpyAsync(
      h_pinned_buffer, d_signal, symbols_per_slot * fft_size * sizeof(hipfftComplex), hipMemcpyDeviceToHost, stream);

  // Wait for all operations to complete
  hipStreamSynchronize(stream);

  // Copy final output back to host
  for (uint32_t i = 0; i < symbols_per_slot; i++) {
    // Copy the result to OFDM symbols
    memcpy(ofdm_symbols + i * nof_sc, h_pinned_buffer + i * fft_size + fft_size - half_subc, half_subc * sizeof(cf_t));
    memcpy(ofdm_symbols + i * nof_sc + half_subc, h_pinned_buffer + i * fft_size, half_subc * sizeof(cf_t));
  }
}